#include "hip/hip_runtime.h"
// C with some CUDA extensions.

__shared__ float2 array1[];
__shared__ __device__ dim3 array2[];
__device__ volatile int i1 = 1;
__constant__ int i2 = 2;

__device__ void dev1(float * a,const float * __restrict__ b)
{
}

__global__ void kernel1(float * a,float * __restrict__ b)
{
	
}

int main()
{
	kernel1<<<1024 * 1024 / 256,256>>>(NULL,NULL);
}

